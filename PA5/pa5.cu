#include "hip/hip_runtime.h"
/*
 * Don Pham - phamd
 * Ryan Davis - davisr3
 */
#include <stdio.h>
#include <stdlib.h>
#include <limits.h>
#include <hip/hip_runtime.h>
#include "ppmFile.h"

__device__ int clamp(int value, int min, int max) {
    return (value < min) ? min : ((value > max) ? max : value);
}

__global__ void blur(int world_size, int blurRadius, int sectionWidth, int sectionHeight,
                     int remainderRows, unsigned char *cleanImageData, unsigned char *blurredImageData)
{
    int id = (blockIdx.z * gridDim.x * gridDim.y + blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
    int imageHeight = sectionHeight * world_size + remainderRows;
    int imageWidth = sectionWidth;

    int sectionByteSize = sectionWidth * sectionHeight * 3;

    // For the rest of the processes
    unsigned char *cleanImagePtr = NULL;
    unsigned char *cleanImageEndPtr = cleanImageData + imageWidth * imageHeight * 3;

    // Pointer to the beginning of each process's unpadded section
    cleanImagePtr = cleanImageData + id * sectionByteSize; // id was 'i' in the for loop

    // paddedHeight is clamped so that it doesn't pass the absolute image bounds
    int rowsAbove = clamp((cleanImagePtr - cleanImageData) / 3 / sectionWidth, 0, INT_MAX);
    int rowsBelow = clamp((cleanImageEndPtr - cleanImagePtr - sectionByteSize) / 3 / sectionWidth, 0, INT_MAX);
    int paddedHeight = sectionHeight
                    + clamp(rowsAbove, 0, blurRadius)
                    + clamp(rowsBelow, 0, blurRadius + ((id == world_size - 1) ? remainderRows : 0));

    // Shift the pointer for the above-padding
    cleanImagePtr -= sectionWidth * clamp(rowsAbove, 0, blurRadius) * 3;

    // Pointers for each process to work with
    unsigned char *cleanSection = cleanImagePtr;
    unsigned char *blurredSection = blurredImageData + id * sectionByteSize;

    // Adjust sectionHeight for the last process, after calculating bounds
    if (id == world_size - 1) {
        sectionHeight += remainderRows;
    }

    printf("Hello world from %i. Section width is: %i, Section height is: %i. Padded height is %i. First pixel is (%u, %u, %u). \n", id, sectionWidth, sectionHeight, paddedHeight, cleanImagePtr[0], cleanImagePtr[1], cleanImagePtr[2]);

    // Do work on the image sections
    int topPaddingOffset = clamp(rowsAbove, 0, blurRadius);
    int bottomPaddingOffset = sectionHeight + topPaddingOffset;

    for (int row = topPaddingOffset; row < bottomPaddingOffset; ++row) {
        for (int col = 0; col < sectionWidth; ++col) {
            // Bounds
            int minX = clamp(col - blurRadius, 0, sectionWidth);
            int maxX = clamp(col + blurRadius, 0, sectionWidth);
            int minY = clamp(row - blurRadius, 0, paddedHeight);
            int maxY = clamp(row + blurRadius, 0, paddedHeight);

            // For each channel (r,g,b)
            for (int channel = 0; channel < 3; ++channel) { // TODO: flip loops for performance
                int sum = 0;
                int numPixels = 0;
                // Take average of pixels
                for (int y = minY; y <= maxY; ++y) {
                    for (int x = minX; x <= maxX; ++x) {
                        sum += cleanSection[(y * sectionWidth + x) * 3 + channel];
                        numPixels += 1;
                    }
                }
                sum = clamp(sum/numPixels, 0, 255);
                // Write average into output
                blurredSection[((row - topPaddingOffset) * sectionWidth + col) * 3 + channel] = sum;
            }
        }
    }
}

int main(int argc, char** argv) {
    int world_size = 10;
    // Initialize the MPI environment
    /*MPI_Init(&argc, &argv);

    // Get the number of processes
    int world_size;
    MPI_Comm_size(MPI_COMM_WORLD, &world_size);

    // Get the rank of the process
    int world_rank;
    MPI_Comm_rank(MPI_COMM_WORLD, &world_rank);

    // Get the name of the processor
    char processor_name[MPI_MAX_PROCESSOR_NAME];
    int name_len;
    MPI_Get_processor_name(processor_name, &name_len);

    // Print off a hello world message
    printf("Hello world from processor %s, rank %d"
           " out of %d processors\n",
           processor_name, world_rank, world_size);
    */
    // Command-line arguments
    int blurRadius = strtol(argv[1], NULL, 10);
    char *inputFile = argv[2];
    char *outputFile = argv[3];

    // Variables set only in the root process
    Image *cleanImage = NULL;
    Image *blurredImage = NULL;
    unsigned char *cleanImageData = NULL; // Pointer to cleanImage->data
    unsigned char *blurredImageData = NULL; // Pointer to blurredImage->data
    int remainderRows = 0;

    // For gatherv
    int *rcounts = NULL; // Size of each section.
    int *displs = NULL; // Displacement of each section.

    // Variables passed from the root process
    int sectionWidth; // The section width (without padding).
    int sectionHeight; // The section height (without padding).
    int sendByteSize; // Number of bytes for the section (including padding).

    // Variables set in each process
    int sectionByteSize; // The size in bytes of the section (without padding).
    int paddedHeight; // The section height after padding the image.
    Image *cleanSection = NULL;
    Image *blurredSection = NULL;

    //if (world_rank == 0)
    //{
    cleanImage = ImageRead(inputFile);
    blurredImage = ImageCreate(cleanImage->width, cleanImage->height);
    cleanImageData = cleanImage->data;
    blurredImageData = blurredImage->data;

    sectionWidth = cleanImage->width;
    sectionHeight = cleanImage->height / world_size;
    remainderRows = cleanImage->height % world_size;
/*
    // Set up sizes for gatherv
    rcounts = (int *) malloc(world_size * sizeof(int));
    displs = (int *) malloc(world_size * sizeof(int));
    for (int i = 0; i < world_size; i++) {
      rcounts[i] = sectionWidth * sectionHeight * 3;
        displs[i] = i * rcounts[i];
    } // The last section is potentially larger than the rest
    rcounts[world_size-1] = sectionWidth * (sectionHeight + remainderRows) * 3;

    // Send parameters
    for (int i = 1; i < world_size; i++) {
        if (i == world_size - 1) { // Last process gets the remainder rows
            sectionHeight += remainderRows;
        }
        //MPI_Send(&sectionWidth, 1, MPI_INT, i, 0, MPI_COMM_WORLD);
        //MPI_Send(&sectionHeight, 1, MPI_INT, i, 0, MPI_COMM_WORLD);
    }
    // Reset sectionHeight for the root process
    sectionHeight -= remainderRows;
*/
    unsigned char *cleanImageDataDevice = NULL;
    unsigned char *blurredImageDataDevice = NULL;
    (hipMalloc ((void **) &cleanImageDataDevice, sizeof(unsigned char) * 3 * cleanImage->width * cleanImage->height));

    (hipMalloc ((void **) &blurredImageDataDevice, sizeof(unsigned char) * 3 * cleanImage->width * cleanImage->height));
    (hipMemcpy(cleanImageDataDevice, cleanImageData, sizeof(unsigned char) * 3 * cleanImage->width * cleanImage->height, hipMemcpyHostToDevice));

    (hipMemcpy(blurredImageDataDevice, blurredImageData, sizeof(unsigned char) * 3 * cleanImage->width * cleanImage->height, hipMemcpyHostToDevice));
    blur<<<world_size, 1>>>(world_size, blurRadius, sectionWidth, sectionHeight, remainderRows, cleanImageDataDevice, blurredImageDataDevice);
    (hipMemcpy(blurredImageData, blurredImageDataDevice, sizeof(unsigned char) * 3 * cleanImage->width * cleanImage->height, hipMemcpyDeviceToHost));

    (hipDeviceSynchronize());

    ImageWrite(blurredImage, outputFile);

    /*}
    else
    {
        MPI_Recv(&sectionWidth, 1, MPI_INT, 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
        MPI_Recv(&sectionHeight, 1, MPI_INT, 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
    }

    // Process variables
    sectionByteSize = sectionWidth * sectionHeight * 3;
    blurredSection = ImageCreate(sectionWidth, sectionHeight);

    // Scatter the image
    if (world_rank == 0)
    {
        // For root process
        paddedHeight = clamp(sectionHeight + ((world_size == 1) ? 0 : blurRadius), 0, cleanImage->height);
        cleanSection = ImageCreate(sectionWidth, paddedHeight);
        free(cleanSection->data);
        cleanSection->data = cleanImageData; // Directly use cleanImageData

        // For the rest of the processes
        unsigned char *cleanImagePtr = NULL;
        unsigned char *cleanImageEndPtr = cleanImageData + cleanImage->width * cleanImage->height * 3;

        for (int i = 1; i < world_size; i++) {
            // Pointer to the beginning of each process's unpadded section
            cleanImagePtr = cleanImageData + i * sectionByteSize;

            // paddedHeight is clamped so that it doesn't pass the absolute image bounds
            int rowsAbove = clamp((cleanImagePtr - cleanImageData) / 3 / sectionWidth, 0, INT_MAX);
            int rowsBelow = clamp((cleanImageEndPtr - cleanImagePtr - sectionByteSize) / 3 / sectionWidth, 0, INT_MAX);
            paddedHeight = sectionHeight
                            + clamp(rowsAbove, 0, blurRadius)
                            + clamp(rowsBelow, 0, blurRadius + ((i == world_size - 1) ? remainderRows : 0));

            // Shift the pointer for the above-padding
            cleanImagePtr -= sectionWidth * clamp(rowsAbove, 0, blurRadius) * 3;

            // Calculate the size that the process will recieve
            sendByteSize = sectionWidth * paddedHeight * 3;

            MPI_Send(&sendByteSize, 1, MPI_INT, i, 0, MPI_COMM_WORLD);
            MPI_Send(cleanImagePtr, sendByteSize, MPI_UNSIGNED_CHAR, i, 1, MPI_COMM_WORLD);
        }
    }
    else
    {
        MPI_Recv(&sendByteSize, 1, MPI_INT, 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
        paddedHeight = sendByteSize / sectionWidth / 3;
        cleanSection = ImageCreate(sectionWidth, paddedHeight); // Allocate space for the data
        MPI_Recv(cleanSection->data, sendByteSize, MPI_UNSIGNED_CHAR, 0, 1, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
    }

    // Gather
    MPI_Gatherv(blurredSection->data, sectionByteSize, MPI_UNSIGNED_CHAR, blurredImageData, rcounts, displs, MPI_UNSIGNED_CHAR, 0, MPI_COMM_WORLD);

    // Write output to file
    if (world_rank == 0) {
        ImageWrite(blurredImage, outputFile);
    }

    // Clean up
    if (world_rank == 0) {
        free(cleanImage->data);
        free(cleanImage);
        free(blurredImage->data);
        free(blurredImage);
        free(cleanSection); // cleanSection->data was already freed
        free(blurredSection->data);
        free(blurredSection);
        free(rcounts);
        free(displs);
    } else {
        free(cleanSection->data);
        free(cleanSection);
        free(blurredSection->data);
        free(blurredSection);
    }

    // Finalize the MPI environment.
    MPI_Finalize();*/
    return 1;
}
