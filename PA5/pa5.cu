#include "hip/hip_runtime.h"
/*
 * Don Pham - phamd
 * Ryan Davis - davisr3
 */
#include <stdio.h>
#include <stdlib.h>
#include <limits.h>
#include <hip/hip_runtime.h>
#include "ppmFile.h"

//This error checking macro is from http://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define CUDA_CHECK_ERROR(response) { check((response), __FILE__, __LINE__); }
inline void check(hipError_t returnCode, const char *file, int line)
{
   if (returnCode != hipSuccess) 
   {
       fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(returnCode), file, line);
       exit(returnCode);
   }
}

__device__ int clamp(int value, int min, int max) {
    return (value < min) ? min : ((value > max) ? max : value);
}

__global__ void blur(int world_size, int blurRadius, int sectionWidth, int sectionHeight,
                     int remainderRows, unsigned char *cleanImageData, unsigned char *blurredImageData)
{
    int id = (blockIdx.z * gridDim.x * gridDim.y + blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
    int imageHeight = sectionHeight * world_size + remainderRows;
    int imageWidth = sectionWidth;

    int sectionByteSize = sectionWidth * sectionHeight * 3;

    // For the rest of the processes
    unsigned char *cleanImagePtr = NULL;
    unsigned char *cleanImageEndPtr = cleanImageData + imageWidth * imageHeight * 3;

    // Pointer to the beginning of each process's unpadded section
    cleanImagePtr = cleanImageData + id * sectionByteSize; // id was 'i' in the for loop

    // paddedHeight is clamped so that it doesn't pass the absolute image bounds
    int rowsAbove = clamp((cleanImagePtr - cleanImageData) / 3 / sectionWidth, 0, INT_MAX);
    int rowsBelow = clamp((cleanImageEndPtr - cleanImagePtr - sectionByteSize) / 3 / sectionWidth, 0, INT_MAX);
    int paddedHeight = sectionHeight
                    + clamp(rowsAbove, 0, blurRadius)
                    + clamp(rowsBelow, 0, blurRadius + ((id == world_size - 1) ? remainderRows : 0));

    // Shift the pointer for the above-padding
    cleanImagePtr -= sectionWidth * clamp(rowsAbove, 0, blurRadius) * 3;

    // Pointers for each process to work with
    unsigned char *cleanSection = cleanImagePtr;
    unsigned char *blurredSection = blurredImageData + id * sectionByteSize;

    // Adjust sectionHeight for the last process, after calculating bounds
    if (id == world_size - 1) {
        sectionHeight += remainderRows;
    }

    printf("Hello world from %i!\n", id);

    // Do work on the image sections
    int topPaddingOffset = clamp(rowsAbove, 0, blurRadius);
    int bottomPaddingOffset = sectionHeight + topPaddingOffset;

    for (int row = topPaddingOffset; row < bottomPaddingOffset; ++row) {
        for (int col = 0; col < sectionWidth; ++col) {
            // Bounds
            int minX = clamp(col - blurRadius, 0, sectionWidth);
            int maxX = clamp(col + blurRadius, 0, sectionWidth);
            int minY = clamp(row - blurRadius, 0, paddedHeight);
            int maxY = clamp(row + blurRadius, 0, paddedHeight);

            // For each channel (r,g,b)
            for (int channel = 0; channel < 3; ++channel) { // TODO: flip loops for performance
                int sum = 0;
                int numPixels = 0;
                // Take average of pixels
                for (int y = minY; y <= maxY; ++y) {
                    for (int x = minX; x <= maxX; ++x) {
                        sum += cleanSection[(y * sectionWidth + x) * 3 + channel];
                        numPixels += 1;
                    }
                }
                sum = clamp(sum/numPixels, 0, 255);
                // Write average into output
                blurredSection[((row - topPaddingOffset) * sectionWidth + col) * 3 + channel] = sum;
            }
        }
    }
}

int main(int argc, char** argv) {
    int world_size = 32;
    // Command-line arguments
    int blurRadius = strtol(argv[1], NULL, 10);
    char *inputFile = argv[2];
    char *outputFile = argv[3];

    // Variables set only in the root process
    Image *cleanImage = NULL;
    Image *blurredImage = NULL;
    unsigned char *cleanImageData = NULL; // Pointer to cleanImage->data
    unsigned char *blurredImageData = NULL; // Pointer to blurredImage->data
    int remainderRows = 0;

    // Variables passed from the root process
    int sectionWidth; // The section width (without padding).
    int sectionHeight; // The section height (without padding).

    cleanImage = ImageRead(inputFile);
    blurredImage = ImageCreate(cleanImage->width, cleanImage->height);
    cleanImageData = cleanImage->data;
    blurredImageData = blurredImage->data;

    sectionWidth = cleanImage->width;
    sectionHeight = cleanImage->height / world_size;
    remainderRows = cleanImage->height % world_size;
    unsigned char *cleanImageDataDevice = NULL;
    unsigned char *blurredImageDataDevice = NULL;
    CUDA_CHECK_ERROR(hipMalloc ((void **) &cleanImageDataDevice, sizeof(unsigned char) * 3 * cleanImage->width * cleanImage->height));

    CUDA_CHECK_ERROR(hipMalloc ((void **) &blurredImageDataDevice, sizeof(unsigned char) * 3 * cleanImage->width * cleanImage->height));
    CUDA_CHECK_ERROR(hipMemcpy(cleanImageDataDevice, cleanImageData, sizeof(unsigned char) * 3 * cleanImage->width * cleanImage->height, hipMemcpyHostToDevice));

    CUDA_CHECK_ERROR(hipMemcpy(blurredImageDataDevice, blurredImageData, sizeof(unsigned char) * 3 * cleanImage->width * cleanImage->height, hipMemcpyHostToDevice));
    blur<<<world_size, 1>>>(world_size, blurRadius, sectionWidth, sectionHeight, remainderRows, cleanImageDataDevice, blurredImageDataDevice);
    CUDA_CHECK_ERROR(hipMemcpy(blurredImageData, blurredImageDataDevice, sizeof(unsigned char) * 3 * cleanImage->width * cleanImage->height, hipMemcpyDeviceToHost));

    CUDA_CHECK_ERROR(hipDeviceSynchronize());

    ImageWrite(blurredImage, outputFile);


    // Clean up
    CUDA_CHECK_ERROR(hipFree ((void *) cleanImageDataDevice));
    CUDA_CHECK_ERROR(hipFree ((void *) blurredImageDataDevice));
    free(cleanImage->data);
    free(cleanImage);
    free(blurredImage->data);
    free(blurredImage);

    return 1;
}
